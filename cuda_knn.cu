#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <thrust/sort.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/host_vector.h>
#include "cuda_knn.h"

#define MAXATTRSIZE 8
#define TRAIN_SIZE 16
#define TEST_SIZE 16
/*
ComputeDistance: Compute the distances between test instances and training instances.
				Save the distance in device_distances

*/
__global__ void kernelComputeDistance(double *trainAttr, double *testAttr, 
	double* device_distances, int trainSize, int testSize, int attrSize){

	// __shared__ double trainData[MAXATTRSIZE * TRAIN_SIZE];//Number of attributes X Number of Train instances in this batch
	// __shared__ double testData[MAXATTRSIZE * TEST_SIZE];//Number of attributes X Number of Test instances in this batch

	int trainIdx = threadIdx.x;
	int testIdx = threadIdx.y;

	int trainOffset = blockDim.x * blockIdx.x;
	int testOffset = blockDim.y * blockIdx.y;

	trainOffset += threadIdx.x;
	testOffset += threadIdx.y;

	//Each thread compute a distance of x to y.


    //Read train data
    //Threads that need the same train instance will read it together
    if(trainOffset < trainSize && testOffset < testSize){
    	double distance = 0.f;
    	for(int i = 0;i < attrSize;i++){
    		double trainAttribute = trainAttr[trainOffset * attrSize + i];

		double testAttribute = testAttr[testOffset * attrSize + i];
		distance += pow(trainAttribute - testAttribute, 2);
		}
		device_distances[testOffset * trainSize + trainOffset] = sqrt(distance);
	}
}


__global__ void initializeIndex(int *device_index, int trainSize, int testSize){
	int testOffset = blockDim.y * blockIdx.y;
	int trainOffset = blockDim.x * blockIdx.x;

	testOffset += threadIdx.y;
	trainOffset += threadIdx.x;
	if(trainOffset < trainSize && testOffset < testSize){
		device_index[testOffset * trainSize + trainOffset] = trainOffset;
	}
}
int *cuPredict(double *trainAttr, int* trainLabels, int trainSize, 
	double *testAttr, int testSize, int attrSize, int k){
	double *device_trainAttr, *device_testAttr, *device_trainLabels, *device_distances;
	int *device_index;

	hipMalloc((void **)&device_trainAttr, sizeof(double) * trainSize * attrSize);
	hipMalloc((void **)&device_trainLabels, sizeof(int) * trainSize);
	hipMalloc((void **)&device_index, sizeof(int) * trainSize * testSize);
	hipMalloc((void **)&device_testAttr, sizeof(double) * testSize * attrSize);
	hipMalloc((void **)&device_distances, sizeof(double) * trainSize * testSize);

	hipMemcpy(device_trainAttr, trainAttr, sizeof(double) * trainSize * attrSize, hipMemcpyHostToDevice);
	hipMemcpy(device_trainLabels, trainLabels, sizeof(int) * trainSize, hipMemcpyHostToDevice);
	hipMemcpy(device_testAttr, testAttr, sizeof(double) * testSize * attrSize, hipMemcpyHostToDevice);

	int blockdimY = (testSize + TEST_SIZE - 1) / TEST_SIZE;
	int blockdimX = (trainSize + TRAIN_SIZE - 1) / TRAIN_SIZE;

	dim3 gridDim(blockdimX, blockdimY);
	dim3 blockDim(TRAIN_SIZE, TEST_SIZE); 


	kernelComputeDistance<<<gridDim, blockDim>>>(device_trainAttr, device_testAttr, 
		device_distances, trainSize,testSize, attrSize);

	initializeIndex<<<gridDim, blockDim>>>(device_index, trainSize,testSize);
	hipDeviceSynchronize();

        thrust::device_ptr<double> keys(device_distances);
        thrust::device_ptr<int> vals(device_index);
	
	for(int i = 0;i < testSize;i++){
//                 printf("here!\n");
		thrust::sort_by_key(keys + i * trainSize, keys + (i + 1) * trainSize, vals + i * trainSize);		
	}

	double *h_distances = new double[trainSize * testSize];
	int *h_idx = new int[trainSize * testSize];

        device_distances = thrust::raw_pointer_cast(keys);
	hipMemcpy(h_distances, device_distances, sizeof(double) * trainSize * testSize, hipMemcpyDeviceToHost);

        for(int i = 0;i < attrSize;i++){
//		printf("attr: %lf\n", testAttr[i]);

	}
	for(int i = 0;i < trainSize;i++){
		printf("%lf\n",h_distances[i]);
	}	

	hipFree(device_distances);

	hipFree(device_index);
	hipFree(device_trainAttr);
	hipFree(device_testAttr);
	hipFree(device_trainLabels);

	int *ret = new int[10];
	return ret;
	//Get distance
	//Sort distance
	//find nearest neighbor
	//return labels
}
