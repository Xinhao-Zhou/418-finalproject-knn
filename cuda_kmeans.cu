#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "cuda_kmeans.h"
#include "exclusiveScan.cu_inl"

#include <stdio.h>

#define BLOCK_DIM 256
#define MAXATTRSIZE 8
#define MAX_K 30
#define ITERATION_THRESHOLD 300
#define ERROR_THRESHOLD 1e-3

//Init a cuda kmeans
cudaKmeans *cudaKmeansInit(int k, int attributesCount, int trainSize, double *trainSet){
	cudaKmeans *ret = new cudaKmeans();

	hipMalloc((void **)&(ret->clusters), sizeof(cudaCluster) * k);

	hipMalloc((void **)&(ret->pointClusterIdx), sizeof(int) * trainSize);

	for(int i = 0;i < k; i++){
		hipMalloc((void **)&(ret->clusters[i].device_attributes), sizeof(double) * trainSize * attributesCount);
		hipMalloc((void **)&(ret->clusters[i].centralPoint), sizeof(double) * attributesCount);
		hipMalloc((void **)&(ret->clusters[i].oldCentralPoint), sizeof(double) * attributesCount);

		hipMemset(&(ret->clusters[i].size), 0, sizeof(int));
		hipMemset(&(ret->clusters[i].attributesCount), attributesCount, sizeof(int));
	}


    srand(k);
    int *randIdxList = new int[k];
    //Get initial central points
	for(int i = 0; i < k; i++){
		int r = rand() % trainSize;
		for(int j = 0;j < i;j++){
			if(randIdxList[j] == r){
				//duplicate random number
				i--;
				break;
			}

			if(j == i - 1){
				randIdxList[i] = r;
				//Copy initial central point.
				hipMemcpy(ret->clusters[i].centralPoint, trainSet + r, sizeof(double) * attributesCount, hipMemcpyHostToDevice);
			}
		}
	}

	delete [] randIdxList;
	return ret;
}

double *MoveTrainSetToCuda(double *trainSet, int trainSize, int attributesCount){
	double *ret;

	hipMalloc((void **)&(ret), sizeof(double) * trainSize * attributesCount);
	hipMemcpy(ret, trainSet, sizeof(double) * trainSize * attributesCount, hipMemcpyHostToDevice);

	return ret;
}

__device__ int getError(double **oldCentralPoints, double **newCentralPoints, int k, int attribtuesCount){

}

__device__ void kmeansIter(){

}

__device__ double distanceFunc(double *attr1, double *attr2, int attributesCount){
	double distance = 0;
	for(int i = 0;i < attributesCount;i++){
		distance += pow(attr1[i] - attr2[i], 2);
	}
	distance = sqrt(distance);
	return distance;
}

__global__ void firstComputeDistance(cudaKmeans *kmeans, double *device_trainSet, int k, int trainSize, int attributesCount){

	__shared__ double centralPoints[MAX_K * MAXATTRSIZE];
	int idx = BLOCK_DIM * blockIdx.x + threadIdx.x;
	double attr[MAXATTRSIZE];
	double minDistance = INFINITY;
	int minIdx = 0;

	if(threadIdx.x < k){
		for(int i = 0;i < attributesCount;i++){
			centralPoints[MAXATTRSIZE * threadIdx.x + i] = kmeans->clusters[threadIdx.x].centralPoint[i];
		}
	}
     
    __syncthreads();
	
	for(int i = 0;i < attributesCount;i++){
		//Read initial attributes
		attr[i] = device_trainSet[idx + i];
	}

	for(int i = 0;i < k;i++){
		double distance = distanceFunc(attr, centralPoints + i * MAXATTRSIZE, attributesCount);
		if(distance < minDistance){
			minDistance = distance;
			minIdx = i;
		}
	}

	__syncthreads();
	//Write cluster idx to global memory.
	kmeans->pointClusterIdx[BLOCK_DIM * blockIdx.x + threadIdx.x] = minIdx;
}

//
__global__ void KmeansUpdateCentralPointsAttributes(cudaKmeans *kmeans, double *device_trainSet, int k, int trainSize, int attributesCount){
	__shared__ double sumArray[BLOCK_DIM * MAXATTRSIZE];
	__shared__ double sumOutput[BLOCK_DIM];
	__shared__ double sumScratch[BLOCK_DIM * 2];

	__shared__ uint inClusterFlag[BLOCK_DIM];
	__shared__ uint inClusterOutput[BLOCK_DIM];
	__shared__ uint inClusterScratch[BLOCK_DIM * 2];

//	__shared__ double oldCentralPoint[MAXATTRSIZE * MAX_K];
//	__shared__ double newCentralPoint[MAXATTRSIZE * MAX_K];

    int pointIdx = blockIdx.x * blockDim.x + threadIdx.x;

    int clusterIdx = kmeans->pointClusterIdx[pointIdx];


    //Set central points' attributes to 0. Store original central points.
    if(threadIdx.x < MAXATTRSIZE * MAX_K){
    	int tmpCId = threadIdx.x / MAXATTRSIZE;
    	int attrID = threadIdx.x % MAXATTRSIZE;

    	//oldCentralPoint[tmpCId * MAXATTRSIZE + attrID] = kmeans->clusters[tmpCId].centralPoint[attrID];
    	kmeans->clusters[tmpCId].centralPoint[attrID] = 0.f;//Set the original central point to 0
   	}

   	//Set cluster size to 0.
   	if(threadIdx.x < k){
   		kmeans->clusters[threadIdx.x].size = 0;
   	}

   	__syncthreads();

    for(int i = 0;i < k;i++){
    	if(i == clusterIdx){
    		inClusterFlag[threadIdx.x] = 1;
    	}else{
    		inClusterFlag[threadIdx.x] = 0;
    	}

    	__syncthreads();
    	//Do prefix sum
    	sharedMemExclusiveScanInt(threadIdx.x, inClusterFlag, inClusterOutput, inClusterScratch, BLOCK_DIM);

    	if(threadIdx.x == BLOCK_DIM - 1){
    		//Add cluster size
    		inClusterOutput[threadIdx.x] += inClusterFlag[threadIdx.x];
    		atomicAdd(&(kmeans->clusters[i].size), inClusterOutput[threadIdx.x]);//Remember to set this to 0!
    	}
    }

	for(int i = 0;i < k; i++){
		if(i == clusterIdx){
			for(int j = 0;j < attributesCount;j++){
				sumArray[BLOCK_DIM * j + threadIdx.x] = device_trainSet[(pointIdx + threadIdx.x) * attributesCount + j];		
			}
		}else{
			for(int j = 0;j < attributesCount;j++){
				sumArray[BLOCK_DIM * j + threadIdx.x] = 0.f;		
			}
		}

		__syncthreads();
		//Sum all attributes inside this block
		for(int j = 0;j < attributesCount;j++){
                        double tmp;
			//Save the last one before the prefix sum.
			if(threadIdx.x == BLOCK_DIM - 1){
				tmp = sumArray[(j + 1) * BLOCK_DIM - 1];
			}

			sharedMemExclusiveScan(threadIdx.x, sumArray + j * BLOCK_DIM, sumOutput, sumScratch, BLOCK_DIM);

			if(threadIdx.x == BLOCK_DIM - 1){
				//Add the last element
				sumOutput[threadIdx.x] += tmp;

				//Add to global variable
				atomicAdd(&(kmeans->clusters[i].centralPoint[j]), sumOutput[threadIdx.x]);
			}
			__syncthreads();
		}
	}
}

__global__ void KmeansGetNewCentralPoint(cudaKmeans *kmeans, int k, int attributesCount){
	__shared__ int sizes[MAX_K];
	__shared__ double newCentralPoints[MAX_K * MAXATTRSIZE];

	if(threadIdx.x < k){
		sizes[threadIdx.x] = kmeans->clusters[threadIdx.x].size;
	}

	__syncthreads();

	if(threadIdx.x < k * attributesCount){
		int row = threadIdx.x / attributesCount;
		int col = threadIdx.x % attributesCount;

		newCentralPoints[row * attributesCount + col] = kmeans->clusters[row].centralPoint[col] / sizes[row];
		kmeans->clusters[row].centralPoint[col] = newCentralPoints[row * attributesCount + col];//write back
	}

}

__global__ void compareOldAndNewCentralPoint(cudaKmeans *kmeans, int *quitFlag, int iteration, int k, int attributesCount){
	if(threadIdx.x == 0){
		*quitFlag = 0;
	}

	__syncthreads();

	if(iteration > ITERATION_THRESHOLD){
		*quitFlag = 1;
		return;
	}

	if(threadIdx.x < k * attributesCount){ 
		int row = threadIdx.x / attributesCount;
		int col = threadIdx.x % attributesCount;

		double diff = fabs(kmeans->clusters[row].centralPoint[col] - kmeans->clusters[row].oldCentralPoint[col]);
		if(diff > ERROR_THRESHOLD){
			atomicAdd(quitFlag, 1);
		}
		kmeans->clusters[row].oldCentralPoint[col] = kmeans->clusters[row].centralPoint[col];//Set old central point.
	}
}

cudaKmeans *getClusters(double *trainSet, int trainSize, int attributesCount, int k){
	double *device_trainSet;

	cudaKmeans *kmeans = cudaKmeansInit(k, attributesCount, trainSize, trainSet);
	device_trainSet = MoveTrainSetToCuda(trainSet, trainSize, attributesCount);

	int blockCount = (trainSize + BLOCK_DIM - 1) / BLOCK_DIM;
	int *device_quitFlag, quitFlag;
	int iteration = 0;
	quitFlag = 0;
	
	hipMalloc((void**)&device_quitFlag, sizeof(int));

	firstComputeDistance<<<blockCount, BLOCK_DIM>>>(kmeans, device_trainSet, k, trainSize, attributesCount);

	for(;quitFlag != 0;iteration++){
		KmeansUpdateCentralPointsAttributes<<<blockCount, BLOCK_DIM>>>(kmeans, device_trainSet, k, trainSize, attributesCount);
		hipDeviceSynchronize();
		KmeansGetNewCentralPoint<<<1, BLOCK_DIM>>>(kmeans, k, attributesCount);
		compareOldAndNewCentralPoint<<<1, BLOCK_DIM>>>(kmeans, device_quitFlag, iteration, k, attributesCount);
		hipMemcpy(&quitFlag, device_quitFlag, sizeof(int), hipMemcpyDeviceToHost);
	}
}
