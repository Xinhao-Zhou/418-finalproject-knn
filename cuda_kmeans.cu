#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "cuda_kmeans.h"
#include "exclusiveScan.cu_inl"

#include <stdio.h>

#define BLOCK_DIM 256
#define MAXATTRSIZE 8
#define MAX_K 30
#define ITERATION_THRESHOLD 300
#define ERROR_THRESHOLD 1e-4

//Init a cuda kmeans
cudaKmeans *cudaKmeansInit(int k, int attributesCount, int trainSize, double *trainSet){
	cudaKmeans *ret = new cudaKmeans();

	hipMalloc((void **)&(ret->clusters), sizeof(cudaCluster) * k);

	hipMalloc((void **)&(ret->pointClusterIdx), sizeof(int) * trainSize);

	for(int i = 0;i < k; i++){
		hipMalloc((void **)&(ret->clusters[i].device_attributes), sizeof(double) * trainSize * attributesCount);
		hipMalloc((void **)&(ret->clusters[i].centralPoint), sizeof(double) * attributesCount);
		hipMalloc((void **)&(ret->clusters[i].oldCentralPoint), sizeof(double) * attributesCount);

		hipMemset(&(ret->clusters[i].size), 0, sizeof(int));
		hipMemset(&(ret->clusters[i].attributesCount), attributesCount, sizeof(int));
	}


    srand(k);
    int *randIdxList = new int[k];
    //Get initial central points
	for(int i = 0; i < k; i++){
		int r = rand() % trainSize;
		for(int j = 0;j < i;j++){
			if(randIdxList[j] == r){
				//duplicate random number
				i--;
				break;
			}

			if(j == i - 1){
				randIdxList[i] = r;
				//Copy initial central point.
				hipMemcpy(ret->clusters[i].centralPoint, trainSet + r, sizeof(double) * attributesCount, hipMemcpyHostToDevice);
			}
		}
	}

	delete [] randIdxList;
	return ret;
}

double *MoveTrainSetToCuda(double *trainSet, int trainSize, int attributesCount){
	double *ret;

	hipMalloc((void **)&(ret), sizeof(double) * trainSize * attributesCount);
	hipMemcpy(ret, trainSet, sizeof(double) * trainSize * attributesCount, hipMemcpyHostToDevice);

	return ret;
}

__device__ int getError(double **oldCentralPoints, double **newCentralPoints, int k, int attribtuesCount){

}

__device__ void kmeansIter(){

}

__device__ double distanceFunc(double *attr1, double *attr2, int attributesCount){
	double distance = 0;
	for(int i = 0;i < attributesCount;i++){
		distance += pow(attr1[i] - attr2[i], 2);
	}
	distance = sqrt(distance);
	return distance;
}

__global__ void firstComputeDistance(double *centralPoint, int *pointClusterIdx, double *device_trainSet, int k, int trainSize, int attributesCount){

	__shared__ double centralPoints[MAX_K * MAXATTRSIZE];
	int idx = BLOCK_DIM * blockIdx.x + threadIdx.x;
	double attr[MAXATTRSIZE];
	double minDistance = INFINITY;
	int minIdx = 0;

	if(threadIdx.x < k){
		for(int i = 0;i < attributesCount;i++){
			centralPoints[MAXATTRSIZE * threadIdx.x + i] = centralPoint[attributesCount * threadIdx.x + i];
		}
	}
     
    __syncthreads();
        if(idx < trainSize){	
	for(int i = 0;i < attributesCount;i++){
		//Read initial attributes
		attr[i] = device_trainSet[idx * attributesCount + i];
	}

	for(int i = 0;i < k;i++){
		double distance = distanceFunc(attr, centralPoints + i * MAXATTRSIZE, attributesCount);
		if(distance < minDistance){
			minDistance = distance;
			minIdx = i;
		}
	}
        }
	__syncthreads();
	//Write cluster idx to global memory.
	pointClusterIdx[BLOCK_DIM * blockIdx.x + threadIdx.x] = minIdx;
}

//
__global__ void KmeansUpdateCentralPointsAttributes(int iteration, double *centralPoint, int *clusterSize, int *pointClusterIdx, double *device_trainSet, int k, int trainSize, int attributesCount){
	__shared__ double sumArray[BLOCK_DIM * MAXATTRSIZE];
	__shared__ double sumOutput[BLOCK_DIM];
	__shared__ double sumScratch[BLOCK_DIM * 2];

	__shared__ uint inClusterFlag[BLOCK_DIM];
	__shared__ uint inClusterOutput[BLOCK_DIM];
	__shared__ uint inClusterScratch[BLOCK_DIM * 2];

//	__shared__ double oldCentralPoint[MAXATTRSIZE * MAX_K];
//	__shared__ double newCentralPoint[MAXATTRSIZE * MAX_K];
    int pointIdx = blockIdx.x * blockDim.x + threadIdx.x;
 
    int clusterIdx = -1;
    if(pointIdx < trainSize)clusterIdx = pointClusterIdx[pointIdx];

    //Set central points' attributes to 0. Store original central points.
    if(threadIdx.x < MAXATTRSIZE * MAX_K){
    	int tmpCId = threadIdx.x / MAXATTRSIZE;
    	int attrID = threadIdx.x % MAXATTRSIZE;

    	//oldCentralPoint[tmpCId * MAXATTRSIZE + attrID] = kmeans->clusters[tmpCId].centralPoint[attrID];
    	centralPoint[attrID] = 0.f;//Set the original central point to 0
   	}

   	//Set cluster size to 0.
   	if(threadIdx.x < k){
   		clusterSize[threadIdx.x] = 0;
   	}

   	__syncthreads();

    for(int i = 0;i < k;i++){

        inClusterOutput[2 * threadIdx.x] = 0;
        inClusterScratch[2 * threadIdx.x] = 0;
        inClusterScratch[2 * threadIdx.x + 1] = 0;
    	if(i == clusterIdx){
    		inClusterFlag[threadIdx.x] = 1;
    	}else{
    		inClusterFlag[threadIdx.x] = 0;
    	}

    	__syncthreads();
    	//Do prefix sum
    	sharedMemExclusiveScanInt(threadIdx.x, inClusterFlag, inClusterOutput, inClusterScratch, BLOCK_DIM);

    	if(threadIdx.x == BLOCK_DIM - 1){
    		//Add cluster size
    		inClusterOutput[threadIdx.x] += inClusterFlag[threadIdx.x];
    		atomicAdd(&(clusterSize[i]), inClusterOutput[threadIdx.x]);//Remember to set this to 0!
    	}
    }
	for(int i = 0;i < k; i++){
         sumOutput[threadIdx.x] = 0;
         sumScratch[2 * threadIdx.x] = 0;    
         sumScratch[2 * threadIdx.x + 1] = 0;
		if(i == clusterIdx){
			for(int j = 0;j < attributesCount;j++){
				sumArray[BLOCK_DIM * j + threadIdx.x] = device_trainSet[(pointIdx) * attributesCount + j];		
			}
		}else{
			for(int j = 0;j < attributesCount;j++){
				sumArray[BLOCK_DIM * j + threadIdx.x] = 0.f;		
			}
		}

		__syncthreads();
		//Sum all attributes inside this block
		for(int j = 0;j < attributesCount;j++){
            double tmp;
			//Save the last one before the prefix sum.
			if(threadIdx.x == BLOCK_DIM - 1){
				tmp = sumArray[(j + 1) * BLOCK_DIM - 1];
			}

			sharedMemExclusiveScan(threadIdx.x, sumArray + j * BLOCK_DIM, sumOutput, sumScratch, BLOCK_DIM);
    
			if(threadIdx.x == BLOCK_DIM - 1){
				//Add the last element
				sumOutput[threadIdx.x] += tmp;
				//Add to global variable
				atomicAdd(&(centralPoint[i * attributesCount + j]), sumOutput[threadIdx.x]);
			}
			__syncthreads();
		}
	}
}

__global__ void KmeansGetNewCentralPoint(double *centralPoint, int *clusterSize, int k, int attributesCount){
	__shared__ int sizes[MAX_K];
	__shared__ double newCentralPoints[MAX_K * MAXATTRSIZE];

	if(threadIdx.x < k){
		sizes[threadIdx.x] = clusterSize[threadIdx.x];
	}

	__syncthreads();

	if(threadIdx.x < k * attributesCount){
		int row = threadIdx.x / attributesCount;
		int col = threadIdx.x % attributesCount;

		newCentralPoints[row * attributesCount + col] = centralPoint[row * attributesCount + col] / sizes[row];
		centralPoint[row * attributesCount + col] = newCentralPoints[row * attributesCount + col];//write back
	}	
}

__global__ void compareOldAndNewCentralPoint(double *centralPoint, double *oldCentralPoint, int *quitFlag, int iteration, int k, int attributesCount){
	__shared__ double diffs[MAX_K * MAXATTRSIZE];

	if(threadIdx.x == 0){
		*quitFlag = 0;
	}

	__syncthreads();

	if(iteration > ITERATION_THRESHOLD){
		*quitFlag = 0;
		return;
	}

	if(threadIdx.x < k * attributesCount){ 
		int row = threadIdx.x / attributesCount;
		int col = threadIdx.x % attributesCount;
		int offset = row * attributesCount + col;

		double diff = fabs(centralPoint[offset] - oldCentralPoint[offset]);
                diffs[offset] = diff;
		//if(diff > ERROR_THRESHOLD){
		//	atomicAdd(quitFlag, 1);
		//}
		oldCentralPoint[offset] = centralPoint[offset];//Set old central point.
	}

        if(threadIdx.x < k){
                double distanceTmp = 0.f;
                for(int i = 0;i < attributesCount;i++){
                    distanceTmp += pow(diffs[threadIdx.x * attributesCount + i], 2);
                }
                distanceTmp = sqrt(distanceTmp);
                if(distanceTmp > ERROR_THRESHOLD){
                    atomicAdd(quitFlag, 1);
                }
        }

}

__global__ void getNewClusterCenter(double *trainSet, int k, int attributesCount, int* pointClusterIdx, double *centralPoint, int trainSize){
    __shared__ double centralPoints[MAX_K * MAXATTRSIZE];
    __shared__ double attributes[BLOCK_DIM * MAXATTRSIZE];
    int pointIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if(threadIdx.x < k * attributesCount){
        centralPoints[threadIdx.x] = centralPoint[threadIdx.x];
    }
    __syncthreads();
    double minDistance = INFINITY;
    int minIdx = -1;
    if(pointIdx < trainSize){
    for(int i = 0;i < attributesCount;i++){
        attributes[threadIdx.x * attributesCount + i] = trainSet[pointIdx * attributesCount + i];
    }
    for(int i = 0;i < k;i++){
        double distance = 0.f;
        for(int j = 0;j < attributesCount;j++){
             double tmp = attributes[threadIdx.x * attributesCount + j] - centralPoints[i * attributesCount + j];
    	     distance += pow(tmp, 2);
        }
        distance = sqrt(distance);
        if(distance < minDistance){
            minDistance = distance;
            minIdx = i;
        }
    }
    pointClusterIdx[pointIdx] = minIdx;
    }


}

const cudaKmeans & getClusters(double *trainSet, int trainSize, int attributesCount, int k){
	double *device_trainSet;
	int *pointClusterIdx;
	double *centralPoint;
	double *oldCentralPoint;
	int *clusterSize;

	hipMalloc((void **)&pointClusterIdx, sizeof(int) * trainSize);
	hipMalloc((void **)&centralPoint, sizeof(double) * attributesCount * k);
	hipMalloc((void **)&oldCentralPoint, sizeof(double) * attributesCount * k);
	hipMalloc((void **)&clusterSize, sizeof(int) * k);

	hipMemset(clusterSize, 0, sizeof(int) * k);

	srand(k);

    int *tmpRandList = new int[k];
	for(int i = 0;i < k;i++){
		int tmpRand = rand() % trainSize;
		if(i == 0){
			tmpRandList[i] = tmpRand;
		}
		for(int j = 0;j < i;j++){
			if(tmpRand == tmpRandList[j]){
				i--;
				break;
			}
			if(j == i - 1){
				tmpRandList[i] = tmpRand;
			}
		}
	}
	for(int i = 0;i < k;i++){
		int idx = tmpRandList[i];
		hipMemcpy(centralPoint + i * attributesCount, trainSet + idx * attributesCount,sizeof(double) * attributesCount, hipMemcpyHostToDevice);
		hipMemcpy(oldCentralPoint + i * attributesCount, trainSet+ idx * attributesCount,sizeof(double) * attributesCount, hipMemcpyHostToDevice);
	}
	delete [] tmpRandList;
	device_trainSet = MoveTrainSetToCuda(trainSet, trainSize, attributesCount);

	int blockCount = (trainSize + BLOCK_DIM - 1) / BLOCK_DIM;
	int *device_quitFlag, quitFlag;
	int iteration = 0;
	quitFlag = 1;
	
	hipMalloc((void**)&device_quitFlag, sizeof(int));

	firstComputeDistance<<<blockCount, BLOCK_DIM>>>(centralPoint, pointClusterIdx, device_trainSet, k, trainSize, attributesCount);
	for(;quitFlag > 0;iteration++){
		KmeansUpdateCentralPointsAttributes<<<blockCount, BLOCK_DIM>>>(iteration,centralPoint, clusterSize, pointClusterIdx, device_trainSet, k, trainSize, attributesCount);
		hipDeviceSynchronize();
		KmeansGetNewCentralPoint<<<1, BLOCK_DIM>>>(centralPoint, clusterSize, k, attributesCount);
		compareOldAndNewCentralPoint<<<1, BLOCK_DIM>>>(centralPoint, oldCentralPoint, device_quitFlag, iteration, k, attributesCount);
		hipMemcpy(&quitFlag, device_quitFlag, sizeof(int), hipMemcpyDeviceToHost);
 getNewClusterCenter<<<blockCount, BLOCK_DIM>>>(device_trainSet, k, attributesCount, pointClusterIdx, centralPoint, trainSize);
	}
	//Copy from device to host..


	int *host_clusterSize = new int[k];
	int *host_pointClusterIdx = new int[trainSize];

	hipMemcpy(host_clusterSize, clusterSize, sizeof(int) * k, hipMemcpyDeviceToHost);
	hipMemcpy(host_pointClusterIdx, pointClusterIdx, sizeof(int) * trainSize,  hipMemcpyDeviceToHost);

	cudaKmeans cuRet = new cudaKmeans();
	cuRet.clusters = new clusters[k];

	for(int i = 0;i < k;i++){
		cuRet.clusters[i].attributesCount = attributesCount;
		cuRet.clusters[i].size = host_clusterSize[i];
		hipMemcpy(cuRet.clusters[i].centralPoint, centralPoint + i * attributesCount, sizeof(double) * attributesCount, hipMemcpyDeviceToHost);
	}

	int *clusterIdx = new int[k]();

	for(int i = 0;i < trainSize;i++){
		int idx = host_pointClusterIdx[i];
		for(int j = 0;j < attributesCount;j++){
			//Assign instances to clusters
			cuRet.clusters[idx].attributes[clusterIdx[idx] * attributesCount + j] = trainSet[i * attributesCount + j];			
		}
		clusterIdx[idx]++;

	}

	delete [] clusterSize;
	delete [] clusterIdx;


	hipFree(device_trainSet);
	hipFree(pointClusterIdx);
	hipFree(centralPoint);
	hipFree(oldCentralPoint);
	hipFree(device_trainSet);
	hipFree(device_quitFlag);

	return cuRet;
}
